/*
 * cudavars.cu
 *
 *  Created on: 10 Jan, 2013
 *      Author: zhao
 */
#include "../util5/cudpp.h"
#include "../util5/hip/hip_runtime_api.h"
#include "cudatypes.h"

//corresponds to the cdscan field in sextractor
float			*d_cdPixArray; //cdvalue field

//corresponds to the the scan field in sextractor
float			*d_pixelArray; //dvalue field
int				*d_labelArray;
int				*d_equivArray;
unsigned int	*d_indexArray;

////////////////used to compact before detection
unsigned int	*d_compactMask;
size_t			*d_numPixAboveThresh;

////////////////temp variables, allocated in the first detection pass(except. d_compactedParentLabel)
unsigned int	*d_segmentMask;
unsigned int	*d_compactedIndexArray;
unsigned int	*d_compactedLabelArray;
unsigned int	*d_compactedParentLabel;
float			*d_compactedcdPixelArray;
float			*d_compactedPixelArray;

unsigned int	*d_pixelCountMask;
unsigned int	*d_pixelCountSegment;

float			*d_fdpeakSegment;
float			*d_fdfluxSegment;
unsigned int 	*d_prunedSegmentMask;

/*---------------The output of the first scan-------------*/
unsigned int	*d_pixelCountArray; //fdnpix
float			*d_fdpeakArray;
float			*d_fdfluxArray;
float			*d_dthreshArray;
unsigned int	*d_ok;

unsigned int	*d_finalPixelIndexArray;
unsigned int	*d_finalLabelArray;
unsigned int	*d_finalObjIndexArray;

/*---------------data structure to background result-----------*/
float 			*d_mean;
float 			*d_sigma;


/*-----------data structure to store objects after cutting-----------*/
unsigned int	*d_cuttedObjLevelArray;
unsigned int	*d_cuttedObjLabelArray;
unsigned int	*d_cuttedObjIndexArray;
unsigned int	*d_cuttedObjFlagArray;
unsigned int	*d_cuttedPixCountArray;
unsigned int	*d_cuttedRootlabelArray;
float			*d_cuttedDthreshArray;

/*-----------data structure to store objects attributes-----------*/
unsigned int	*d_index;
unsigned int	*d_flag;
unsigned int	*d_fdnpix;
float			*d_dthresh;

unsigned int 	*d_xmin;
unsigned int 	*d_xmax;
unsigned int 	*d_ymin;
unsigned int 	*d_ymax;
unsigned int 	*d_dnpix;
unsigned int 	*d_npix;
unsigned int	*d_peakx;
unsigned int	*d_peaky;

float 		*d_bkg;
float 		*d_dbkg;
float		*d_sigbkg;
float 		*d_a;
float 		*d_b;
float 		*d_cxx;
float 		*d_cxy;
float 		*d_cyy;
float 		*d_theta;
float 		*d_abcor;
float		*d_peak;
float 		*d_dpeak;
float 		*d_fdpeak;
float		*d_flux;
float 		*d_dflux;
float 		*d_fdflux;
float		*d_fluxerr;
float		*d_thresh;
float		*d_mthresh;
float		*d_fwhm;

double 		*d_mx;
double 		*d_my;
double 		*d_mx2;
double 		*d_my2;
double 		*d_mxy;
double		*d_poserr_mx2;
double		*d_poserr_my2;
double		*d_poserr_mxy;
char  		*d_singuflag;

int			*d_iso[NISO];
//////////////////////////
//used in endobject phase
double		*d_mxw;
double		*d_myw;
double		*d_alphas;
double		*d_deltas;
double		*d_alpha2000;
double		*d_delta2000;

double 		*d_posx;
double		*d_posy;
float		*d_elong;
float		*d_ellip;
float		*d_polar;
float		*d_sprob;
float		*d_sposx;
float		*d_sposy;
//float		*d_poserr_a;
//float		*d_poserr_b;
//float		*d_poserr_theta;
//float		*d_poserr_cxx;
//float		*d_poserr_cyy;
//float		*d_poserr_cxy;
float		*d_flux_iso;
float		*d_fluxerr_iso;
float		*d_flux_isocor;
float		*d_fluxerr_isocor;
float		*d_mag_iso;
float		*d_magerr_iso;
float		*d_kronfactor;
float		*d_flux_auto;
float		*d_fluxerr_auto;
float		*d_mag_auto;
float		*d_magerr_auto;

float		*d_flux_aper[NAPER];
float		*d_fluxerr_aper[NAPER];
float		*d_mag_aper[NAPER];
float		*d_magerr_aper[NAPER];

/*-----------CUDPP variables-----------*/
CUDPPHandle 	theCudpp;

CUDPPConfiguration config;
CUDPPHandle 	scanplan;
CUDPPHandle		reduceplan;
CUDPPHandle		segscanplan;
CUDPPHandle		compactplan;
CUDPPHandle		sortplan;

int 	width;
int 	height;
float	thresh;
float 	global_dthresh;

/*-----------timing variables-----------*/
hipEvent_t start_t, stop_t;	//total time measurement
hipEvent_t start, stop;		//time measurement of each part

unsigned int 	*d_masterIndex;
