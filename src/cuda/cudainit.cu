/*
 * cudainit.cu
 *
 *  Created on: 10 Jan, 2013
 *      Author: zhao
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "../util5/cudpp.h"
#include "../util5/hip/hip_runtime_api.h"
#include "cudatypes.h"


extern "C" void init_device(int _width, int _height, float *imgbuf)
{
	//hipDeviceReset();

	width = _width;
	height = _height;

	hipEventCreate(&start_t);
	hipEventCreate(&stop_t);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start_t, 0);


	checkCudaErrors(hipMalloc((void**)&d_cdPixArray, width*height * sizeof(float)));

	checkCudaErrors(hipMalloc((void**)&d_pixelArray, width*height * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_labelArray, width*height * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&d_equivArray, width*height * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&d_compactMask,width*height * sizeof(unsigned int)));

	checkCudaErrors(hipMalloc((void**)&d_numPixAboveThresh, sizeof(size_t)));


	float time;
	hipEventRecord(start, 0);

	hipMemcpy((float*)d_pixelArray, imgbuf, width*height*sizeof(float), hipMemcpyHostToDevice);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

#ifdef DEBUG_CUDA
	printf("time consumed by cuda data transfer in is %f\n", time);
#endif

	//Initialize the CUDPP Library
	cudppCreate(&theCudpp);

}

void init_objects(int total)
{
	d_index 	= d_cuttedObjLabelArray;
	d_flag 		= d_cuttedObjFlagArray;
	d_fdnpix 	= d_cuttedPixCountArray;
	d_dthresh 	= d_cuttedDthreshArray;

	hipMalloc((void**)&d_xmin, total*sizeof(unsigned int));
	hipMalloc((void**)&d_xmax, total*sizeof(unsigned int));
	hipMalloc((void**)&d_ymin, total*sizeof(unsigned int));
	hipMalloc((void**)&d_ymax, total*sizeof(unsigned int));
	hipMalloc((void**)&d_dnpix,total*sizeof(unsigned int));
	hipMalloc((void**)&d_npix, total*sizeof(unsigned int));
	hipMalloc((void**)&d_peakx,total*sizeof(unsigned int));
	hipMalloc((void**)&d_peaky,total*sizeof(unsigned int));

	hipMalloc((void**)&d_bkg, 		total*sizeof(float));
	hipMalloc((void**)&d_dbkg, 	total*sizeof(float));
	hipMalloc((void**)&d_sigbkg, 	total*sizeof(float));
	hipMalloc((void**)&d_a, 		total*sizeof(float));
	hipMalloc((void**)&d_b, 		total*sizeof(float));
	hipMalloc((void**)&d_cxx,		total*sizeof(float));
	hipMalloc((void**)&d_cxy,		total*sizeof(float));
	hipMalloc((void**)&d_cyy,		total*sizeof(float));
	hipMalloc((void**)&d_theta,	total*sizeof(float));
	hipMalloc((void**)&d_abcor,	total*sizeof(float));
	hipMalloc((void**)&d_peak, 	total*sizeof(float));
	hipMalloc((void**)&d_dpeak, 	total*sizeof(float));
	hipMalloc((void**)&d_fdpeak, 	total*sizeof(float));
	hipMalloc((void**)&d_flux, 	total*sizeof(float));
	hipMalloc((void**)&d_dflux,	total*sizeof(float));
	hipMalloc((void**)&d_fdflux,	total*sizeof(float));
	hipMalloc((void**)&d_fluxerr,	total*sizeof(float));
	hipMalloc((void**)&d_thresh,	total*sizeof(float));
	hipMalloc((void**)&d_mthresh,	total*sizeof(float));
	hipMalloc((void**)&d_fwhm, 	total*sizeof(float));

	hipMalloc((void**)&d_mx, 		total*sizeof(double));
	hipMalloc((void**)&d_my, 		total*sizeof(double));
	hipMalloc((void**)&d_mx2, 		total*sizeof(double));
	hipMalloc((void**)&d_my2, 		total*sizeof(double));
	hipMalloc((void**)&d_mxy, 		total*sizeof(double));
	hipMalloc((void**)&d_poserr_mx2, 	total*sizeof(double));
	hipMalloc((void**)&d_poserr_my2, 	total*sizeof(double));
	hipMalloc((void**)&d_poserr_mxy, 	total*sizeof(double));
	hipMalloc((void**)&d_singuflag,	total*sizeof(char));

	//for astrom measurement
	hipMalloc((void**)&d_mxw, 		total*sizeof(double));
	hipMalloc((void**)&d_myw, 		total*sizeof(double));
	hipMalloc((void**)&d_alphas, 	total*sizeof(double));
	hipMalloc((void**)&d_deltas, 	total*sizeof(double));
	hipMalloc((void**)&d_alpha2000,total*sizeof(double));
	hipMalloc((void**)&d_delta2000,total*sizeof(double));

	//hipMalloc((void**)&d_iso,			NISO*sizeof(int*));
	for(int i=0; i<NISO; i++)
	{
		hipMalloc((void**)&d_iso[i],	total*sizeof(int));
		hipMemset(d_iso[i], 0, total*sizeof(int));
	}

	//for variables used in endobject phase
	hipMalloc((void**)&d_posx, total*sizeof(double));
	hipMalloc((void**)&d_posy, total*sizeof(double));

	hipMalloc((void**)&d_elong, 	total*sizeof(float));
	hipMalloc((void**)&d_ellip, 	total*sizeof(float));
	hipMalloc((void**)&d_polar, 	total*sizeof(float));
	hipMalloc((void**)&d_sprob, 	total*sizeof(float));
	hipMalloc((void**)&d_sposx, 	total*sizeof(float));
	hipMalloc((void**)&d_sposy, 	total*sizeof(float));
	//hipMalloc((void**)&d_poserr_a, total*sizeof(float));
	//hipMalloc((void**)&d_poserr_b, total*sizeof(float));
	//hipMalloc((void**)&d_poserr_theta, total*sizeof(float));
	//hipMalloc((void**)&d_poserr_cxx, total*sizeof(float));
	//hipMalloc((void**)&d_poserr_cyy, total*sizeof(float));
	//hipMalloc((void**)&d_poserr_cxy, total*sizeof(float));
	hipMalloc((void**)&d_flux_iso, total*sizeof(float));
	hipMalloc((void**)&d_fluxerr_iso, total*sizeof(float));
	hipMalloc((void**)&d_flux_isocor, total*sizeof(float));
	hipMalloc((void**)&d_fluxerr_isocor, total*sizeof(float));
	hipMalloc((void**)&d_mag_iso, total*sizeof(float));
	hipMalloc((void**)&d_magerr_iso, total*sizeof(float));
	hipMalloc((void**)&d_kronfactor, total*sizeof(float));
	hipMalloc((void**)&d_flux_auto, total*sizeof(float));
	hipMalloc((void**)&d_fluxerr_auto, total*sizeof(float));
	hipMalloc((void**)&d_mag_auto, total*sizeof(float));
	hipMalloc((void**)&d_magerr_auto, total*sizeof(float));

	for(int i=0; i<NAPER; i++)
	{
		hipMalloc((void**)&d_flux_aper[i],	total*sizeof(float));
		hipMalloc((void**)&d_fluxerr_aper[i],	total*sizeof(float));
		hipMalloc((void**)&d_mag_aper[i],	total*sizeof(float));
		hipMalloc((void**)&d_magerr_aper[i],	total*sizeof(float));
	}

}


extern "C" void clear_device(float *img)
{
	hipMemcpy(img, d_pixelArray, width*height*sizeof(float), hipMemcpyDeviceToHost);

	checkCudaErrors(hipFree(d_cdPixArray));
	checkCudaErrors(hipFree(d_pixelArray));

	checkCudaErrors(hipFree(d_labelArray));
	checkCudaErrors(hipFree(d_equivArray));
	checkCudaErrors(hipFree(d_compactMask));
	checkCudaErrors(hipFree(d_numPixAboveThresh));

	checkCudaErrors(hipFree(d_compactedIndexArray));
	checkCudaErrors(hipFree(d_compactedParentLabel));
	//checkCudaErrors(hipFree(d_compactedPixelArray));
	checkCudaErrors(hipFree(d_compactedcdPixelArray));
	checkCudaErrors(hipFree(d_segmentMask));

	checkCudaErrors(hipFree(d_pixelCountMask));
	checkCudaErrors(hipFree(d_pixelCountSegment));
	checkCudaErrors(hipFree(d_fdpeakSegment));
	checkCudaErrors(hipFree(d_fdfluxSegment));
	checkCudaErrors(hipFree(d_prunedSegmentMask));

	/*-----------data structure to store objects after cutting-----------*/
	//unfinished, need to free all
	checkCudaErrors(hipFree(d_cuttedObjLevelArray));
	checkCudaErrors(hipFree(d_cuttedObjLabelArray));
	checkCudaErrors(hipFree(d_cuttedObjIndexArray));
	checkCudaErrors(hipFree(d_cuttedObjFlagArray));
	checkCudaErrors(hipFree(d_cuttedPixCountArray));
	checkCudaErrors(hipFree(d_cuttedRootlabelArray));
	checkCudaErrors(hipFree(d_cuttedDthreshArray));

	checkCudaErrors(hipFree(d_xmin));
	checkCudaErrors(hipFree(d_xmax));
	checkCudaErrors(hipFree(d_ymin));
	checkCudaErrors(hipFree(d_ymax));
	checkCudaErrors(hipFree(d_dnpix));
	checkCudaErrors(hipFree(d_mx));
	checkCudaErrors(hipFree(d_my));
	checkCudaErrors(hipFree(d_mx2));
	checkCudaErrors(hipFree(d_my2));
	checkCudaErrors(hipFree(d_mxy));
	checkCudaErrors(hipFree(d_cxx));
	checkCudaErrors(hipFree(d_cxy));
	checkCudaErrors(hipFree(d_cyy));
	checkCudaErrors(hipFree(d_a));
	checkCudaErrors(hipFree(d_b));
	checkCudaErrors(hipFree(d_theta));
	checkCudaErrors(hipFree(d_abcor));
	checkCudaErrors(hipFree(d_fdpeak));
	checkCudaErrors(hipFree(d_dpeak));
	checkCudaErrors(hipFree(d_fdflux));
	checkCudaErrors(hipFree(d_dflux));
	checkCudaErrors(hipFree(d_singuflag));

	for(int i=0; i<NISO; i++)
		checkCudaErrors(hipFree(d_iso[i]));

	checkCudaErrors(hipFree(d_posx));
	checkCudaErrors(hipFree(d_posy));
	checkCudaErrors(hipFree(d_elong));
	checkCudaErrors(hipFree(d_ellip));
	checkCudaErrors(hipFree(d_polar));
	checkCudaErrors(hipFree(d_sprob));
	checkCudaErrors(hipFree(d_sposx));
	checkCudaErrors(hipFree(d_sposy));
	checkCudaErrors(hipFree(d_flux_iso));
	checkCudaErrors(hipFree(d_fluxerr_iso));
	checkCudaErrors(hipFree(d_flux_isocor));
	checkCudaErrors(hipFree(d_fluxerr_isocor));
	checkCudaErrors(hipFree(d_mag_iso));
	checkCudaErrors(hipFree(d_magerr_iso));
	checkCudaErrors(hipFree(d_kronfactor));
	checkCudaErrors(hipFree(d_flux_auto));
	checkCudaErrors(hipFree(d_fluxerr_auto));
	checkCudaErrors(hipFree(d_mag_auto));
	checkCudaErrors(hipFree(d_magerr_auto));

	for(int i=0; i<NAPER; i++)
	{
		checkCudaErrors(hipFree(d_flux_aper[i]));
		checkCudaErrors(hipFree(d_fluxerr_aper[i]));
		checkCudaErrors(hipFree(d_mag_aper[i]));
		checkCudaErrors(hipFree(d_magerr_aper[i]));
	}

	//for astrom measurement
	checkCudaErrors(hipFree(d_mxw));
	checkCudaErrors(hipFree(d_myw));
	checkCudaErrors(hipFree(d_alphas));
	checkCudaErrors(hipFree(d_deltas));
	checkCudaErrors(hipFree(d_alpha2000));
	checkCudaErrors(hipFree(d_delta2000));

	/*---------------data structure to background result-----------*/
	checkCudaErrors(hipFree(d_mean));
	checkCudaErrors(hipFree(d_sigma));

	checkCudaErrors(hipFree(d_masterIndex));

	// Shut down the CUDPP library
	cudppDestroy(theCudpp);

	/*-----------------timing variables-------------------*/
	hipEventRecord(stop_t, 0);
	hipEventSynchronize(stop_t);

	float totaltime;
	hipEventElapsedTime(&totaltime, start_t, stop_t);

	hipEventDestroy(start_t);
	hipEventDestroy(stop_t);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Time counsumed by gpu code in SExtractor is: %f (ms)\n", totaltime);

}



